#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <time.h>
#include <string.h>
#include <chrono>
#include "opencv4/opencv2/opencv.hpp"
texture<unsigned char, 2 , hipReadModeElementType> textImg;
__global__ void convolution(
                            uchar *ans,
                            int8_t *mask,uint imgW, uint imgH){
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    if(x > imgW || y > imgH)
        return;
    int16_t total = 0;
    for (uint8_t i = 0 ; i < 3 ; i++){
        for(uint8_t j = 0 ; j < 3 ; j++){
            total += (int16_t)tex2D(textImg, x+i, y+j) * mask[i + j * 3];
        }
    }
    if(total < 0){
        total = 0;
    }
    __syncthreads();
    ans[x  + y * imgW] = total;
}

__global__ void noTextConvolution(uchar *ans, int8_t *mask, uint imgW, uint imgH, uchar * img){
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    if(x > imgW || y > imgH)
        return;
    int16_t total = 0;
    for (uint8_t i = 0 ; i < 3 ; i++){
        for(uint8_t j = 0 ; j < 3 ; j++){
            total += (int16_t)img[(x+i) +  (y+j) * imgW] * mask[i + j * 3];
        }
    }
    if(total < 0){
        total = 0;
    }
    __syncthreads();
    ans[x  + y * imgW] = total;
}

int main() {
    cv::Mat orgImg =  cv::imread("/home/ascdc/Downloads/1280px-RoadEcologyConference2017-17.jpg",cv::IMREAD_GRAYSCALE);
    cv::imshow("G",orgImg);
    unsigned char *ptrImg = orgImg.isContinuous()? orgImg.data: orgImg.clone().data;

    //use hipChannelFormatDesc define struct type
    //hipChannelFormatDesc chDesc = hipCreateChannelDesc(8,8,0,0,hipChannelFormatKindUnsigned);
    //or
    hipChannelFormatDesc chDesc = hipCreateChannelDesc<uchar>();
    hipArray * cuArr;
    hipError_t err;
    err = hipMallocArray(&cuArr, &chDesc, orgImg.cols, orgImg.rows);
    if(err != hipSuccess){
        std::cout << hipGetErrorString(err) << std::endl;
        return -1;
    }
    err = hipMemcpyToArray(cuArr, 0, 0, ptrImg, sizeof(uchar) * orgImg.total(), hipMemcpyHostToDevice);
    if(err != hipSuccess){
        std::cout << hipGetErrorString(err) << std::endl;
        return -1;
    }

    //binding
    hipBindTextureToArray(&textImg, cuArr, &chDesc);

    //alloc mask space
    int8_t mask[9] = {-1,-1,-1,
                      -1,8,-1,
                      -1,-1,-1};
    int8_t* devMask;
    hipMalloc((void**)&devMask, sizeof(int8_t) * 9);
    err = hipMemcpy(devMask, mask, sizeof(int8_t) * 9, hipMemcpyHostToDevice);
    if(err != hipSuccess){
        std::cout << hipGetErrorString(err) << std::endl;
        return -1;
    }


    //alloc output space
    uchar *output = new uchar[orgImg.total()];
    uchar *devOut;
    hipMalloc((void**)&devOut, sizeof(uchar) * orgImg.total());




    dim3 block(orgImg.cols / 32 + 1, orgImg.rows / 32 + 1);
    dim3 thread(32,32,1);
    for(auto i = 0 ; i < 100 ; i++){
        convolution<<<block,thread>>>(devOut, devMask, orgImg.cols, orgImg.rows);

    }
    hipDeviceSynchronize();
    err = hipGetLastError();
    if(err != hipSuccess){
        std::cout << hipGetErrorString(err) << std::endl;
    }
    hipMemcpy(output, devOut, sizeof(uchar) * orgImg.total() , hipMemcpyDeviceToHost);
    hipFree(devOut);
    cv::Mat outMat(orgImg.rows,orgImg.cols,CV_8UC1, (unsigned char*)output);
    cv::imwrite("text.png",outMat);

    delete []output;
    outMat.release();
    hipUnbindTexture(&textImg);
    hipFree(cuArr);





    //test 2
    //without texture
    uchar *devImg;
    hipMalloc((void**)& devImg, sizeof(uchar) * orgImg.total());
    hipMemcpy(devImg, ptrImg, sizeof(uchar) * orgImg.total() , hipMemcpyHostToDevice);
    output = new uchar[orgImg.total()];
    hipMalloc((void**)&devOut, sizeof(uchar) * orgImg.total());
    for(auto i = 0 ; i < 100 ; i++) {

        noTextConvolution<<<block, thread>>>(devOut, devMask, orgImg.cols, orgImg.rows, devImg);
    }
    hipDeviceSynchronize();
    err = hipMemcpy(output, devOut, sizeof(uchar) * orgImg.total() , hipMemcpyDeviceToHost);
    if(err != hipSuccess){
        std::cout << hipGetErrorString(err) << std::endl;
    }
    hipFree(devOut);
    outMat = cv::Mat(orgImg.rows,orgImg.cols,CV_8UC1, (unsigned char*)output);
    cv::imwrite("nontext.png",outMat);

    hipFree(devImg);
    hipFree(devOut);
    delete []output;


    hipFree(devMask);
    orgImg.release();
    return 0;
}
