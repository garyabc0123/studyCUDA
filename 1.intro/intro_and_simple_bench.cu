﻿
#include "hip/hip_runtime.h"





#include <iostream>
#include <stdio.h>
#include <time.h>
#include <chrono>
//#define SIZE 4194303*1024 //1024*1024

size_t SIZE = 131072 * 1024;
#define BLOCKSIZE 1024
__global__ void deviceADD(int* a, int* b, int* c) {
	int off = threadIdx.x + blockIdx.x * blockDim.x;
	c[off] = a[off] + b[off];
}
void fillramdom(size_t size, int* ptr) {
	for (size_t i = 0; i < size; i++) {
		i[ptr] = rand();
	}
}
void errhand(hipError_t err) {
	if (err) {
		printf("Error: %s\n", hipGetErrorString(err));
		std::cout << err << std::endl;
	}
}
//cpu版計算function
std::chrono::duration<double > calByCPU(size_t size, int* a, int* b, int* ans) {
	auto start = std::chrono::high_resolution_clock::now();
	for (size_t i = 0; i < size; i++) {
		i[ans] = i[a] + i[b];
	}
	auto end = std::chrono::high_resolution_clock::now();
	return std::chrono::duration_cast < std::chrono::duration<double >> (end - start);
}
//gpu版計算function
//計時器只計算扣掉搬data後的計算時間
std::chrono::duration<double > calByGPU(size_t size, int* a, int* b, int* ans) {
	int* gpuA, * gpuB, * gpuC;
	hipError_t err;



	auto allocStart = std::chrono::high_resolution_clock::now();
	err = hipMalloc((void**)&gpuA, size * sizeof(int));
	errhand(err);
	err = hipMalloc((void**)&gpuB, size * sizeof(int));
	errhand(err);
	err = hipMalloc((void**)&gpuC, size * sizeof(int));
	errhand(err);

	err = hipMemcpy(gpuA, a, size * sizeof(int), hipMemcpyHostToDevice);
	errhand(err);
	err = hipMemcpy(gpuB, b, size * sizeof(int), hipMemcpyHostToDevice);
	errhand(err);
	auto allocEnd = std::chrono::high_resolution_clock::now();
	auto allocTime = std::chrono::duration_cast <std::chrono::duration<double >> (allocEnd - allocStart);
	std::cout << "Alloc time : " << allocTime.count() << std::endl;



	dim3 gridDim(SIZE / BLOCKSIZE,1,1);
	auto start = std::chrono::high_resolution_clock::now();
	deviceADD <<<gridDim, BLOCKSIZE >>> (gpuA, gpuB, gpuC);
	//<<<block, thread>>>
	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now();

	err = hipGetLastError();
	errhand(err);
	//<<<block, thread>>>
	err = hipMemcpy(ans, gpuC, size * sizeof(int), hipMemcpyDeviceToHost);
	errhand(err);
	hipDeviceSynchronize();
	hipFree(gpuA);
	hipFree(gpuB);
	hipFree(gpuC);
	return std::chrono::duration_cast <std::chrono::duration<double >> (end - start);

}
bool equal(size_t size, int* a, int* b) {
	for (size_t i = 0; i < size; i++) {
		if (i[a] != i[b]) {
			std::cout << i[a] << "\t" << i[b] << "\t" << i << std::endl;
			return false;
		}
		
	}
	return true;
}
void printArr(size_t size, int* ptr) {
	for (size_t i = 0; i < size; i++) {
		std::cout << i << ": " << i[ptr] << std::endl;
	}
}

void benchmark(int time, std::chrono::duration<double > (*func)(size_t, int*, int*, int*), size_t size, int *a , int *b , int * c) {
	std::cout << "start benchmark" << std::endl;
	std::cout << "Time\texecute Time" << std::endl;
	std::cout << "---------------" << std::endl;
	double total = 0;
	for (int i = 0; i < time; i++) {
		auto exeT = (*func)(size, a, b, c);
		std::cout << i + 1 << "\t" << exeT.count() << std::endl;
		total += exeT.count();
	}
	std::cout << "average : " << total / time << std::endl;
	std::cout << "------end------ " << std::endl;

}

int main(void) {
	srand(time(NULL));
	int* a;
	int* b;
	int* c_cpu;
	int* c_gpu;
	a = new int[SIZE];
	b = new int[SIZE];
	c_cpu = new int[SIZE];
	c_gpu = new int[SIZE];
	fillramdom(SIZE, a);
	fillramdom(SIZE, b);
	benchmark(10, calByCPU, SIZE, a, b, c_cpu);
	benchmark(10, calByGPU, SIZE, a, b, c_gpu);

	//calByCPU(SIZE, a, b, c_cpu);
	//calByGPU(SIZE, a, b, c_gpu);
	//printArr(SIZE, c_gpu);
	std::cout << (equal(SIZE, c_cpu, c_gpu) ? "True" : "False") << std::endl;
	delete[] a, b, c_cpu, c_gpu;
}

